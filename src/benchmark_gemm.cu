#include "hip/hip_runtime.h"
// Save this as benchmark_gemm.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// External declarations for the PTX kernels
extern "C" __global__ void gemm4x4(float* A, float* B, float* C);
extern "C" __global__ void gemm4x4_1d_tiled(float* A, float* B, float* C);

// CPU reference implementation for verification
void gemm4x4_cpu(float* A, float* B, float* C) {
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            float sum = 0.0f;
            for (int k = 0; k < 4; k++) {
                sum += A[i*4+k] * B[k*4+j];
            }
            C[i*4+j] = sum;
        }
    }
}

// Utility function to check if results match
bool verify_result(float* expected, float* actual, int size) {
    const float epsilon = 1e-5;
    for (int i = 0; i < size; i++) {
        if (fabs(expected[i] - actual[i]) > epsilon) {
            printf("Mismatch at index %d: expected %f, got %f\n", i, expected[i], actual[i]);
            return false;
        }
    }
    return true;
}

// Benchmark function
void benchmark_kernel(void (*kernel)(float*, float*, float*), float* d_A, float* d_B, float* d_C, 
                      const char* kernel_name, int num_runs) {
    // Warmup
    kernel<<<1, 1>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    
    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    for (int i = a 0; i < num_runs; i++) {
        kernel<<<1, 1>>>(d_A, d_B, d_C);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("%s: average time per run = %f ms (over %d runs)\n", 
           kernel_name, milliseconds / num_runs, num_runs);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    // Allocate host memory
    float *h_A, *h_B, *h_C, *h_C_ref;
    h_A = (float*)malloc(16 * sizeof(float));
    h_B = (float*)malloc(16 * sizeof(float));
    h_C = (float*)malloc(16 * sizeof(float));
    h_C_ref = (float*)malloc(16 * sizeof(float));
    
    // Initialize matrices with random values
    srand(time(NULL));
    for (int i = 0; i < 16; i++) {
        h_A[i] = (float)(rand() % 10);
        h_B[i] = (float)(rand() % 10);
    }
    
    // Compute reference result on CPU
    gemm4x4_cpu(h_A, h_B, h_C_ref);
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, 16 * sizeof(float));
    hipMalloc(&d_B, 16 * sizeof(float));
    hipMalloc(&d_C, 16 * sizeof(float));
    
    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, 16 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, 16 * sizeof(float), hipMemcpyHostToDevice);
    
    // Benchmark and verify naive implementation
    benchmark_kernel(gemm4x4, d_A, d_B, d_C, "Naive GEMM", 1000);
    hipMemcpy(h_C, d_C, 16 * sizeof(float), hipMemcpyDeviceToHost);
    printf("Naive GEMM: %s\n", verify_result(h_C_ref, h_C, 16) ? "PASSED" : "FAILED");
    
    // Benchmark and verify tiled implementation
    benchmark_kernel(gemm4x4_1d_tiled, d_A, d_B, d_C, "1D Tiled GEMM", 1000);
    hipMemcpy(h_C, d_C, 16 * sizeof(float), hipMemcpyDeviceToHost);
    printf("1D Tiled GEMM: %s\n", verify_result(h_C_ref, h_C, 16) ? "PASSED" : "FAILED");
    
    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
}
